#include "hip/hip_runtime.h"
#include <iostream>

#include <ScalarMulRunner.cuh>
#include <CommonKernels.cuh>

/**
 * @property
 * num_elements <= block_size^2
 */
float ScalarMulTwoReductions(int num_elements,
                             float* vector1,
                             float* vector2,
                             int block_size) {
  size_t log_num_elements = Log2(num_elements);

  // Device memory.
  int vec_byte_size = num_elements * sizeof(float);

  float* d_v1;
  hipMalloc(&d_v1, vec_byte_size);

  float* d_v2;
  hipMalloc(&d_v2, vec_byte_size);

  float* d_block_result;
  int reduction_block_count = (num_elements + block_size - 1) / block_size;
  hipMalloc(&d_block_result, reduction_block_count * sizeof(float));

  float* d_result;
  hipMalloc(&d_result, sizeof(float));

  // Get reduced_block_size (2^p, where p: 2^p >= reduction_block_count).
  int p_counter = 1;
  int blocks_left = (reduction_block_count - 1) >> 1;
  while (blocks_left > 0) {
    ++p_counter;
    blocks_left >>= 1;
  }
  int reduced_block_size = 1 << p_counter;

  // Move to device.
  hipMemcpy(d_v1, vector1, vec_byte_size, hipMemcpyHostToDevice);
  hipMemcpy(d_v2, vector2, vec_byte_size, hipMemcpyHostToDevice);

  // Events.
  hipEvent_t start;
  hipEventCreate(&start);

  hipEvent_t stop;
  hipEventCreate(&stop);

  // Function execution.
  hipEventRecord(start);

  ScalarMulPerBlockWarpSpecific<<<
  reduction_block_count, block_size, block_size * sizeof(float)
  >>>(num_elements, d_v1, d_v2, d_block_result);
//  ScalarMulPerBlockSimple<<<
//      reduction_block_count, block_size, block_size * sizeof(float)
//      >>>(num_elements, d_v1, d_v2, d_block_result);

  hipDeviceSynchronize();

  SumElementsInFirstBlock<<<
  1, reduced_block_size, block_size * sizeof(float)
  >>>(reduced_block_size, d_block_result, d_result);

  hipEventRecord(stop);

  float* result = new float[1];
  hipMemcpy(result, d_result, sizeof(float), hipMemcpyDeviceToHost);

  // Put elapsed time.
  hipEventSynchronize(stop);

  cudaErrchk(hipPeekAtLastError());

  float millis = 0;
  hipEventElapsedTime(&millis, start, stop);
  std::cout << "ScalarMulTwoReductions, "
            << block_size << ", "
            << "$2^{" << log_num_elements << "}$, "
            << millis << '\n';

  // Free resources.
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(d_v1);
  hipFree(d_v2);
  hipFree(d_block_result);
  hipFree(d_result);

  float res = result[0];
  delete[] result;

  return res;
}

float ScalarMulSumPlusReduction(int num_elements,
                                float* vector1,
                                float* vector2,
                                int block_size) {
  size_t log_num_elements = Log2(num_elements);

  // Device memory.
  int vec_byte_size = num_elements * sizeof(float);

  float* d_v1;
  hipMalloc(&d_v1, vec_byte_size);

  float* d_v2;
  hipMalloc(&d_v2, vec_byte_size);

  float* d_block_result;
  int d_block_result_size = std::min(block_size, num_elements);
  hipMalloc(&d_block_result, d_block_result_size * sizeof(float));

  float* d_result;
  hipMalloc(&d_result, sizeof(float));

  // Move to device.
  hipMemcpy(d_v1, vector1, vec_byte_size, hipMemcpyHostToDevice);
  hipMemcpy(d_v2, vector2, vec_byte_size, hipMemcpyHostToDevice);

  // Events.
  hipEvent_t start;
  hipEventCreate(&start);

  hipEvent_t stop;
  hipEventCreate(&stop);

  // Function execution.
  hipEventRecord(start);

  ScalarMulSumToOneBlock<<<
  d_block_result_size, block_size, block_size * sizeof(float)
  >>>(num_elements, d_v1, d_v2, d_block_result);

  hipDeviceSynchronize();

  SumElementsInFirstBlock<<<
  1, block_size, block_size * sizeof(float)
  >>>(d_block_result_size, d_block_result, d_result);

  hipEventRecord(stop);

  float* result = new float[1];
  hipMemcpy(result, d_result, sizeof(float), hipMemcpyDeviceToHost);

  cudaErrchk(hipPeekAtLastError());

  // Put elapsed time.
  hipEventSynchronize(stop);

  float millis = 0;
  hipEventElapsedTime(&millis, start, stop);
  std::cout << "ScalarMulSumPlusReduction, "
            << block_size << ", "
            << "$2^{" << log_num_elements << "}$, "
            << millis << '\n';

  // Free resources.
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(d_v1);
  hipFree(d_v2);
  hipFree(d_block_result);
  hipFree(d_result);

  float res = result[0];
  delete[] result;

  return res;
}
