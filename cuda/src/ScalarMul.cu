#include "hip/hip_runtime.h"
#include <ScalarMul.cuh>
#include <CommonKernels.cuh>

// Without profiling data.
float ScalarMul(int num_elements,
                float* vector1,
                float* vector2,
                int block_size) {
  // Device memory.
  int vec_byte_size = num_elements * sizeof(float);

  float* d_v1;
  hipMalloc(&d_v1, vec_byte_size);

  float* d_v2;
  hipMalloc(&d_v2, vec_byte_size);

  float* d_block_result;
  int d_block_result_size = std::min(block_size, num_elements);
  hipMalloc(&d_block_result, d_block_result_size * sizeof(float));

  float* d_result;
  hipMalloc(&d_result, sizeof(float));

  // Move to device.
  hipMemcpy(d_v1, vector1, vec_byte_size, hipMemcpyHostToDevice);
  hipMemcpy(d_v2, vector2, vec_byte_size, hipMemcpyHostToDevice);

  // Function execution.
  ScalarMulSumToOneBlock<<<
  d_block_result_size, block_size, block_size * sizeof(float)
  >>>(num_elements, d_v1, d_v2, d_block_result);

  hipDeviceSynchronize();

  SumElementsInFirstBlock<<<
  1, block_size, block_size * sizeof(float)
  >>>(d_block_result_size, d_block_result, d_result);

  float* result = new float[1];
  hipMemcpy(result, d_result, sizeof(float), hipMemcpyDeviceToHost);

  cudaErrchk(hipPeekAtLastError());

  // Free resources.
  hipFree(d_v1);
  hipFree(d_v2);
  hipFree(d_block_result);
  hipFree(d_result);

  float res = result[0];
  delete[] result;

  return res;
}
